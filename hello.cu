#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize(); // Add this to ensure all CUDA operations complete.
    return 0;
}
